
#include <hip/hip_runtime.h>
extern "C"  __global__ void fill(int * A, int cnt){
	const int gap = blockDim.x*gridDim.x;
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < cnt; id += gap)
		A[id] = id * 2;
};
